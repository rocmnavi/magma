#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.7.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date August 2023

       @generated from magmablas/zaxpycp.cu, normal z -> s, Fri Aug 25 13:18:31 2023

*/
#include "magma_internal.h"

#define NB 64

/******************************************************************************/
// adds   x += r  --and--
// copies r = b
// each thread does one index, x[i] and r[i]
__global__ void
saxpycp_kernel(
    int m,
    float *r,
    float *x,
    const float *b)
{
    const int i = threadIdx.x + blockIdx.x*NB;
    if ( i < m ) {
        x[i] = MAGMA_S_ADD( x[i], r[i] );
        r[i] = b[i];
    }
}


/***************************************************************************//**
    adds   x += r  --and--
    copies r = b
*******************************************************************************/
extern "C" void
magmablas_saxpycp(
    magma_int_t m,
    magmaFloat_ptr r,
    magmaFloat_ptr x,
    magmaFloat_const_ptr b,
    magma_queue_t queue )
{
    dim3 threads( NB );
    dim3 grid( magma_ceildiv( m, NB ) );
    saxpycp_kernel <<< grid, threads, 0, queue->hip_stream() >>> ( m, r, x, b );
}
